#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <nvml.h>
#include <vector>
#include <chrono>

__global__ void mem_copy_kernel(float* dst, const float* src, size_t N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) dst[i] = src[i];
}



void print_memory_utilization() {
    nvmlInit();
    nvmlDevice_t dev;
    nvmlDeviceGetHandleByIndex(0, &dev);
    nvmlUtilization_t utilization;
    nvmlDeviceGetUtilizationRates(dev, &utilization);
    std::cout << "[NVML] GPU Util: " << utilization.gpu << "%, Mem Util: " << utilization.memory << "%\n";
    nvmlShutdown();
}

void stress_h2d_d2h_bandwidth(size_t N, int repetitions) {
    float *h_data, *d_data;
    hipHostMalloc(&h_data, N * sizeof(float), hipHostMallocDefault);  // pinned host memory
    hipMalloc(&d_data, N * sizeof(float));

    for (int i = 0; i < N; ++i) h_data[i] = 1.0f;

    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < repetitions; ++i) {
        hipMemcpy(d_data, h_data, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(h_data, d_data, N * sizeof(float), hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    double total_bytes = 2.0 * repetitions * N * sizeof(float); // H2D + D2H
    double time_sec = std::chrono::duration<double>(end - start).count();
    double bandwidth_GBps = total_bytes / (time_sec * 1e9);

    std::cout << "[H2D+D2H] " << bandwidth_GBps << " GB/s over " << repetitions << " repetitions\n";

    hipHostFree(h_data);
    hipFree(d_data);
}

void stress_d2d_bandwidth(size_t N, int repetitions) {
    float *src, *dst;
    hipMalloc(&src, N * sizeof(float));
    hipMalloc(&dst, N * sizeof(float));
    hipMemset(src, 1, N * sizeof(float));

    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < repetitions; ++i) {
        hipMemcpy(dst, src, N * sizeof(float), hipMemcpyDeviceToDevice);
    }
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    double total_bytes = repetitions * N * sizeof(float);
    double time_sec = std::chrono::duration<double>(end - start).count();
    double bandwidth_GBps = total_bytes / (time_sec * 1e9);

    std::cout << "[D2D] " << bandwidth_GBps << " GB/s over " << repetitions << " repetitions\n";

    hipFree(src);
    hipFree(dst);
}


void stress_async_streamed_h2d_d2h(size_t N, int reps, int num_streams) {
    std::vector<hipStream_t> streams(num_streams);
    float *h_data[num_streams], *d_data[num_streams];

    for (int s = 0; s < num_streams; ++s) {
        hipStreamCreate(&streams[s]);
        hipHostMalloc(&h_data[s], N * sizeof(float), hipHostMallocDefault);
        hipMalloc(&d_data[s], N * sizeof(float));
        std::fill(h_data[s], h_data[s] + N, 1.0f);
    }

    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < reps; ++i) {
        for (int s = 0; s < num_streams; ++s) {
            hipMemcpyAsync(d_data[s], h_data[s], N * sizeof(float), hipMemcpyHostToDevice, streams[s]);
            hipMemcpyAsync(h_data[s], d_data[s], N * sizeof(float), hipMemcpyDeviceToHost, streams[s]);
        }
    }

    for (int s = 0; s < num_streams; ++s) {
        hipStreamSynchronize(streams[s]);
    }

    auto end = std::chrono::high_resolution_clock::now();

    double total_bytes = 2.0 * reps * num_streams * N * sizeof(float);
    double time_sec = std::chrono::duration<double>(end - start).count();
    double bandwidth = total_bytes / (time_sec * 1e9);

    std::cout << "[Async H2D+D2H] " << bandwidth << " GB/s with " << num_streams << " streams\n";

    for (int s = 0; s < num_streams; ++s) {
        hipStreamDestroy(streams[s]);
        hipFree(d_data[s]);
        hipHostFree(h_data[s]);
    }
}


void stress_nvlink_peer_copy(size_t N, int reps) {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount < 2) {
        std::cerr << "[NVLink] Skipping peer test — only one GPU found.\n";
        return;
    }

    hipSetDevice(0);
    float* d0; hipMalloc(&d0, N * sizeof(float));
    hipSetDevice(1);
    float* d1; hipMalloc(&d1, N * sizeof(float));

    hipDeviceEnablePeerAccess(0, 0);
    hipDeviceEnablePeerAccess(1, 0);

    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < reps; ++i) {
        hipMemcpyPeer(d0, 0, d1, 1, N * sizeof(float));
        hipMemcpyPeer(d1, 1, d0, 0, N * sizeof(float));
    }
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    double total_bytes = 2.0 * reps * N * sizeof(float);
    double time_sec = std::chrono::duration<double>(end - start).count();
    double bandwidth = total_bytes / (time_sec * 1e9);

    std::cout << "[NVLink P2P] " << bandwidth << " GB/s\n";

    hipFree(d0);
    hipFree(d1);
}

void run_stress_test() {
    size_t N = 1 << 24;
    float *d_src, *d_dst;
    hipMalloc(&d_src, N * sizeof(float));
    hipMalloc(&d_dst, N * sizeof(float));

    auto start = std::chrono::high_resolution_clock::now();
    mem_copy_kernel<<<(N + 255) / 256, 256>>>(d_dst, d_src, N);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    std::ofstream log("logs/stress_log.csv", std::ios::app);
    std::cout << "memory," << N << "," << std::chrono::duration<double, std::milli>(end - start).count() << std::endl;
    // fflush(stdout);
    log.close();
    

    std::cout << "Memory stress test completed." << std::endl;
    hipFree(d_src);
    hipFree(d_dst);

    N = 1 << 26; // 64MB
    const int reps = 100;

    std::cout << "--- Running Bandwidth Stress Tests ---\n";
    stress_h2d_d2h_bandwidth(N, reps);
    std::cout << "--- Running D2D Bandwidth Stress Test ---\n";
    stress_d2d_bandwidth(N, reps);
    std::cout << "--- Running Async Streamed H2D+D2H Stress Test ---\n";
    stress_async_streamed_h2d_d2h(N, reps, 40);
    std::cout << "--- Running NVLink Peer Copy Stress Test ---\n";
    stress_nvlink_peer_copy(N, 100*reps);

    print_memory_utilization();
}
